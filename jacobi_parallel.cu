#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <ctime>
#include <ratio>
#include <chrono>
using namespace std;
#define CONV_THRESHOLD 1e-3

bool check_convergence(int N, double* D, double* D_new){
	double sqr_diff = 0;
	for(int i=0; i<N; i++){
		double diff = D_new[i*N + i] - D[i*N + i];
		if(diff<0) sqr_diff -= diff;
		else sqr_diff += diff;
	}
	cout << sqr_diff << endl;
	return (sqr_diff < CONV_THRESHOLD);
}

double check_eigenvals(int N, double* D){
	double out = 0;
	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++){
			if(i!=j){
				out+=fabs(D[i*N + j]);
			}
		}
	} 
	return out;
}
__global__ void pq_change(int N, int *p, int*q){
	// printf("Updating PQ\n");
	int tid = threadIdx.x;
	int i = blockIdx.x;

	int ind1 = (tid + i)%(N-1);
   	int ind2;
   	if(tid != 0) ind2 = ((N-tid)+i - 1)%(N-1);
   	else ind2 = N - 1;

   	int valp = min(ind1, ind2);
   	int valq = max(ind1, ind2);

   	p[i*(N)/2 + tid] = valp;
   	q[i*(N)/2 + tid] = valq;
}

__global__ void cosandsin(int *N, double *D, double *c, double *s, int *pcurr, int *qcurr){
	// printf("Inside cossin\n");
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	// printf("%d\n", tid);
	int row = pcurr[tid];
	int col = qcurr[tid];

	double p = D[row*(*N) + col];
    double y = (D[col*(*N) + col] - D[row*(*N) + row]) / 2.0;
    double d = fabs(y) + sqrt(p*p + y*y);
    double r = sqrt(p*p + d*d);

    if(fabs(p) < CONV_THRESHOLD && fabs(d) < CONV_THRESHOLD){
    	c[tid] = 1.0;
    	s[tid] = 0.0;
    }
    else{
	    c[tid] = d / r;
		s[tid] = (fabs(y)/y)*(p / r);
    }

}


__global__ void rotate_rows(int* N, double* D, double* out, double* c, double* s, int* pcurr, int* qcurr){
	__shared__ int p, q;
	__shared__ double co, si;

	if(threadIdx.x == 0){
		p = pcurr[blockIdx.x];
		q = qcurr[blockIdx.x];
		co = c[blockIdx.x];
		si = s[blockIdx.x];
		// printf("[@] Inside Row update: \n" );
		// printf("%d %d %f %f \n", p, q, co, si);
	
	}



	__syncthreads();



	int i = threadIdx.x;


	double val1 = D[p*(*N)+i];
	double val2 = D[q*(*N)+i];



	out[i*(*N)+p] = co*val1 - si*val2;

}

__global__ void rotate_rows2(int* N, double* D, double* out, double* c, double* s, int* pcurr, int* qcurr){
	__shared__ int p, q;
	__shared__ double co, si;

	if(threadIdx.x == 0){
		p = pcurr[blockIdx.x];
		q = qcurr[blockIdx.x];
		co = c[blockIdx.x];
		si = s[blockIdx.x];
	}
	__syncthreads();

	int i = threadIdx.x;

	double val1 = D[p*(*N)+i];
	double val2 = D[q*(*N)+i];

	out[i*(*N)+q] = si*val1 + co*val2;

}

__global__ void rotate_cols(int* N, double* D, double* out, double* c, double* s, int* pcurr, int* qcurr){
	__shared__ int p, q;
	__shared__ double co, si;

	if(threadIdx.x == 0){
		p = pcurr[blockIdx.x];
		q = qcurr[blockIdx.x];
		co = c[blockIdx.x];
		si = s[blockIdx.x];
	}
	__syncthreads();
	int i = threadIdx.x;

	double val1 = D[p*(*N)+i];
	double val2 = D[q*(*N)+i];

	out[p*(*N)+i] = co*val1 - si*val2;
	out[q*(*N)+i] = si*val1 + co*val2;

}


__global__ void get_ev(double* old_arr, double* new_arr){
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	new_arr[tid] = old_arr[tid];
}


void jacobi_parallel(int N, double* D, double* eigenvecs_out, double* eigenvals_out){
	double *eigenvals = eigenvals_out;
	double *eigenvecs = eigenvecs_out;
	int N2 = N;
	double *ENEW;
	double *DNEW;
	if(N%2==1){

		ENEW = (double*)calloc((N+1)*(N+1), sizeof(double));
		DNEW = (double*)calloc((N+1)*(N+1), sizeof(double));
		for(int i=0; i<N; i++){
			for(int j=0; j<N; j++){
				DNEW[i*(N+1) + j] = D[i*N + j];
				ENEW[i*(N+1) + j] = 0;
			}
			ENEW[i*(N+1)+i] = 1;
		}
		D = DNEW;
		N = N+1;
		D[N*N - 1] = 1;
		ENEW[N*N - 1] = 1;
		eigenvals = D;
		eigenvecs = ENEW;
		// printf("Done\n");
	}

	double *dD, *Dtemp, *eignevecs_D, *eignevecs_D_temp;
	std::chrono::high_resolution_clock::time_point t1, t2;
	t1 = std::chrono::high_resolution_clock::now();
	hipMalloc((void**)&dD, sizeof(double)*N*N);
	hipMalloc((void**)&Dtemp, sizeof(double)*N*N);
	hipMalloc((void**)&eignevecs_D, sizeof(double)*N*N);
	hipMalloc((void**)&eignevecs_D_temp, sizeof(double)*N*N);

	hipMemcpy(dD, D, sizeof(double)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(Dtemp, D, sizeof(double)*N*N, hipMemcpyHostToDevice);
	
	hipMemcpy(eignevecs_D, eigenvecs, sizeof(double)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(eignevecs_D_temp, eigenvecs, sizeof(double)*N*N, hipMemcpyHostToDevice);

	int *dN, *dp, *dq;
	double *c, *s;
	hipMalloc((void **)&dN, sizeof(int));
	hipMalloc((void **)&c, sizeof(double)*N/2);
	hipMalloc((void **)&s, sizeof(double)*N/2);

	hipMemcpy(dN, &N, sizeof(int), hipMemcpyHostToDevice);

	double *Dvoidtemp = (double*)malloc(sizeof(double)*N*N);
	double conv = false;
	t2 = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> time_span = std::chrono::duration_cast<std::chrono::duration<double>>(t2 - t1);

	hipMalloc((void **)&dp, sizeof(int)*N*(N-1)/2);
	hipMalloc((void **)&dq, sizeof(int)*N*(N-1)/2);
	pq_change<<<N-1, N/2>>>(N, dp, dq);


	int *p = (int*)malloc(sizeof(int)*N*(N-1)/2);
	int *q = (int*)malloc(sizeof(int)*N*(N-1)/2);

	hipMemcpy(p, dp, sizeof(int)*N*(N-1)/2, hipMemcpyDeviceToHost);
	hipMemcpy(q, dq, sizeof(int)*N*(N-1)/2, hipMemcpyDeviceToHost);
	

	hipDeviceSynchronize(); 
	int sweeps = 0;
	while(!conv){
		t1 = std::chrono::high_resolution_clock::now();

		int N1;

		if(N%2 != 0) N1 = N;
		else N1 = N-1;
		for(int i=0; i<N1; i++){
			int *currp = dp+(i*(N/2));
			int *currq = dq+(i*(N/2));

			cossin<<<N/2, 1>>>(dN, dD, c, s, currp, currq);
			hipDeviceSynchronize();
			rotate_rows<<<N/2, N>>>(dN, dD, Dtemp, c, s, currp, currq);
			rotate_rows2<<<N/2, N>>>(dN, dD, Dtemp, c, s, currp, currq);

			hipDeviceSynchronize();

			rotate_cols<<<N/2, N>>>(dN, Dtemp, dD, c, s, currp, currq);

			rotate_cols<<<N/2, N>>>(dN, eignevecs_D, eignevecs_D_temp, c, s, currp, currq);
			hipDeviceSynchronize();
			get_ev<<<N, N>>>(eignevecs_D_temp, eignevecs_D);
			hipDeviceSynchronize();

		}

		hipMemcpy(Dvoidtemp, dD, sizeof(double)*N*N, hipMemcpyDeviceToHost);


		cout << "Sweep " << ++sweeps << ": ";

		conv = check_convergence(N, eigenvals, Dvoidtemp);
		t2 = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double> time_span = std::chrono::duration_cast<std::chrono::duration<double>>(t2 - t1);


		double* tempor = eigenvals;
		eigenvals = Dvoidtemp;
		Dvoidtemp = tempor;
		double valdiff = check_eigenvals(N, eigenvals);

	}

	double* eigenvecs_temp = (double*)malloc(sizeof(double)*N*N);
	hipMemcpy(eigenvecs_temp, eignevecs_D, sizeof(double)*N*N, hipMemcpyDeviceToHost);
	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++){
			eigenvecs[j*N+i] = eigenvecs_temp[N*i + j];
		}
	}

	if(N2%2 == 1){
		for(int i=0; i<N-1; i++){
			for(int j=0; j<N-1; j++){
				eigenvecs_out[i*(N-1) + j] = eigenvecs[i*N + j];
				eigenvals_out[i*(N-1) + j] = eigenvals[i*N + j];

			}
		}
		free(eigenvals);
		free(eigenvecs);
	}

	free(eigenvecs_temp);

	hipFree(dD);
	hipFree(Dtemp);
	hipFree(dN);
	hipFree(c);
	hipFree(s);
	hipFree(dp);
	hipFree(dq);
	hipFree(eignevecs_D);
	hipFree(eignevecs_D_temp);
}
int main(){
	ofstream ofile;
	ofile.open("output.txt");
	hipEvent_t start,end;
	for(int size=2;size<=512;size*=2){
		hipEventCreate(&start);
        hipEventCreate(&end);	
		fstream infile;
		infile.open("input_"+to_string(size)+".txt");
		int N;
		infile>>N;
		hipEventRecord(start);
		double* D=(double*)calloc(N*N, sizeof(double));
		double data;

		double* temp = (double*)calloc(N*N, sizeof(double));
		double* E = (double*)calloc(N*N, sizeof(double));
		
		for(int i=0; i<N; i++){
			E[i*N+i] = 1;
		}
		for(int i=0; i<N; i++){
			infile>>data;
			D[i*N+i] = data;
		}
		
		jacobi_parallel(N,D,E,temp);

		infile.close();

		hipDeviceSynchronize();
        hipEventRecord(end);
        hipEventSynchronize(end);
        float milliseconds=0;
		hipEventElapsedTime(&milliseconds,start,end);
		ofile<<size<<":"<<milliseconds*1000<<"\n";
		}
	
		ofile.close();
		return 0;
	
	
}
